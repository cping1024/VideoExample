#include "hip/hip_runtime.h"
/*
 * NV12ToARGB color space conversion CUDA kernel
 *
 * This sample uses CUDA to perform a simple NV12 (YUV 4:2:0 planar)
 * source and converts to output in ARGB format
 */
//#ifdef HAVE_NVCUVID

#include <util/colorspace.h>
#include <stdio.h>

static int divUp(int m, int n) {
	return (m+n-1) / n;
}

__constant__ float constHueColorSpaceMat[9] = { 1.1644f, 0.0f,1.596f, 1.1644f, -0.3918f, -0.813f, 1.1644f, 2.0172f, 0.0f };
__device__ static void YUV2RGB(const uint* yuvi, float* red, float* green, float* blue)
{
	float luma, chromaCb, chromaCr;
    // Prepare for hue adjustment
	luma     = (float)yuvi[0];
	chromaCb = (float)((int)yuvi[1] - 512.0f);
	chromaCr = (float)((int)yuvi[2] - 512.0f);
    // Convert YUV To RGB with hue adjustment
	*red   = (luma     * constHueColorSpaceMat[0]) +
		(chromaCb * constHueColorSpaceMat[1]) +
		(chromaCr * constHueColorSpaceMat[2]);
       
	*green = (luma     * constHueColorSpaceMat[3]) +
		(chromaCb * constHueColorSpaceMat[4]) +
		(chromaCr * constHueColorSpaceMat[5]);

    *blue  = (luma     * constHueColorSpaceMat[6]) +
		(chromaCb * constHueColorSpaceMat[7]) +
		(chromaCr * constHueColorSpaceMat[8]);
}

__device__ static uint RGBA_pack_10bit(float red, float green, float blue, uint alpha)
{
	uint ARGBpixel = 0;
	// Clamp final 10 bit results
	red   = ::fmin(::fmax(red,   0.0f), 1023.f);
	green = ::fmin(::fmax(green, 0.0f), 1023.f);
	blue  = ::fmin(::fmax(blue,  0.0f), 1023.f);
	// Convert to 8 bit unsigned integers per color component
	ARGBpixel = (((uint)blue  >> 2) |
			(((uint)green >> 2) << 8)  |
			(((uint)red   >> 2) << 16) |
			(uint)alpha);
	return ARGBpixel;
}

// CUDA kernel for outputing the final BGRA output from NV12
#define COLOR_COMPONENT_BIT_SIZE 10
#define COLOR_COMPONENT_MASK     0x3FF
__global__ void nv12_to_bgra_kernel(const unsigned char* in, size_t inpitch, uint* out, size_t outpitch, uint width, uint height)
{
	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	const int y = blockIdx.y *  blockDim.y       +  threadIdx.y;
	if (x >= width || y >= height)
		return;
	// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
	// if we move to texture we could read 4 luminance values
	uint yuv101010Pel[2];
	yuv101010Pel[0] = (in[y * inpitch + x    ]) << 2;
	yuv101010Pel[1] = (in[y * inpitch + x + 1]) << 2;
	const size_t chromaOffset = inpitch * height;
	const int y_chroma = y >> 1;
        
	if (y & 1) {  // odd scanline ?
		uint chromaCb = in[chromaOffset + y_chroma * inpitch + x    ];
		uint chromaCr = in[chromaOffset + y_chroma * inpitch + x + 1];
		if (y_chroma < ((height >> 1) - 1)) {// interpolate chroma vertically
			chromaCb = (chromaCb + in[chromaOffset + (y_chroma + 1) * inpitch + x    ] + 1) >> 1;
			chromaCr = (chromaCr + in[chromaOffset + (y_chroma + 1) * inpitch + x + 1] + 1) >> 1;
        }
        yuv101010Pel[0] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
        yuv101010Pel[1] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    } else {
		yuv101010Pel[0] |= ((uint)in[chromaOffset + y_chroma * inpitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[0] |= ((uint)in[chromaOffset + y_chroma * inpitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
		yuv101010Pel[1] |= ((uint)in[chromaOffset + y_chroma * inpitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[1] |= ((uint)in[chromaOffset + y_chroma * inpitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	}
	// this steps performs the color conversion
	uint yuvi[6];
	float red[2], green[2], blue[2];
	yuvi[0] =  (yuv101010Pel[0] &   COLOR_COMPONENT_MASK    );
	yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
	yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);
	yuvi[3] =  (yuv101010Pel[1] &   COLOR_COMPONENT_MASK    );
	yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
	yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);
	// YUV to RGB Transformation conversion
	YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
	YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);
	// Clamp the results to RGBA
	const size_t pitch = outpitch >> 2;
	out[y * pitch + x     ] = RGBA_pack_10bit(red[0], green[0], blue[0], ((uint)0xff << 24));
	out[y * pitch + x + 1 ] = RGBA_pack_10bit(red[1], green[1], blue[1], ((uint)0xff << 24));
    
}

void nv12_to_bgra(int width, int height, unsigned char* in, int inpitch, unsigned char* out, int outpitch)
{
    // Final Stage: NV12toARGB color space conversion
    dim3 block(32, 8);
    dim3 grid(divUp(width, 2 * block.x), divUp(height, block.y));
    nv12_to_bgra_kernel<<<grid, block>>>(in, inpitch, (unsigned int*)out, outpitch, width, height);
}


__global__ void bgra_to_yuv_kernel(const unsigned char* in, unsigned char* out, int height, int width)
{
}
void bgra_to_yuv(const unsigned char* in, unsigned char* out, int height, int width) 
{
}

__global__ void bgra_to_bgr_kernel(const unsigned char* in, unsigned char* out, int height, int width)
{
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;
	if (tidx < width && tidy < height) {
		out[tidy*width*3 + tidx*3 + 0] = in[tidy*width*4 + tidx*4 + 0];
		out[tidy*width*3 + tidx*3 + 1] = in[tidy*width*4 + tidx*4 + 1];
		out[tidy*width*3 + tidx*3 + 2] = in[tidy*width*4 + tidx*4 + 2];
	}
}

void bgra_to_bgr(const unsigned char* in, unsigned char* out, int height, int width)
{
	dim3 block(16, 16);
	dim3 grid(divUp(width, block.x), divUp(height, block.y));
	bgra_to_bgr_kernel<<<grid, block>>>(in, out, height, width);
}

//#endif
